#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
// #include <windows.h>

#include<unistd.h>

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)tp.tv_sec * 1000 + (double)tp.tv_usec/1000;
}

int recursiveReduce(int *idata, int const size){
    // terminate check
    if (size == 1) return idata[0];

    //renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; ++i){
        idata[i] += idata[stride + i];
    }
    
    //call recursively
    return recursiveReduce(idata, stride);
}

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n){
    //set thread ID
    unsigned int tid = threadIdx.x;

    //convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x*blockDim.x;

    //boundary check
    if(tid >= n) return;

    //in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2){
        if((tid % (2*stride)) == 0){
            idata[tid] += idata[tid + stride];
        }
        //synchronize within threadblock
        __syncthreads();
    }
    //write result for this block to global mem
    if(tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n){
    //set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    //convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x*blockDim.x;

    //boundary check
    if(idx >= n) return;

    //in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2){
        int index = 2*stride*tid;

        if(index < blockDim.x){
            idata[index] += idata[index + stride];
        }
        //synchronize within threadblock
        __syncthreads();
    }
    //write result for this block to global mem
    if(tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;
    // boundary check
    if(idx >=n) return;

    //in-place reduction in global memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1){
        if(tid < stride){
            idata[tid] += idata[tid + stride];
        }
        //synchronize within threadblock
        __syncthreads();
    }

    //write result for this block to global mem
    if(tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n){
    //set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x*2 + threadIdx.x;

    //convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x*blockDim.x*2;

    //unrolling 2
    if(idx + blockDim.x < n)
        g_idata[idx] += g_idata[idx + blockDim.x];
    __syncthreads();

    //in-place reduction in global memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1){
        if(tid < stride){
            idata[tid] += idata[tid + stride];
        }
        //synchronize within threadblock
        __syncthreads();
    }

    //write result for this block to global mem
    if(tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n){
    //set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x*8 + threadIdx.x;

    //convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x*blockDim.x*8;

    //unrolling 2
    if(idx + 7*blockDim.x < n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2*blockDim.x];
        int a4 = g_idata[idx + 3*blockDim.x];
        int b1 = g_idata[idx + 4*blockDim.x];
        int b2 = g_idata[idx + 5*blockDim.x];
        int b3 = g_idata[idx + 6*blockDim.x];
        int b4 = g_idata[idx + 7*blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    __syncthreads();

    //in-place reduction and complete unroll
    if(blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    if(blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    if(blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    if(blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();



    //unrolling warp
    if(tid < 32){
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    //write result for this block to global mem
    if(tid == 0)
        g_odata[blockIdx.x] = idata[0];
}


int main(int argc, char *argv[]) {
    //set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    bool bResult = false;
    
    
    //initialization
    int size = 1<<24;
    printf("Matrix size: %d\n", size);

    //execution configuration
    int blocksize = 512;
    if(argc > 1){
        blocksize = atoi(argv[1]);
    }

    dim3 block (blocksize, 1);
    dim3 grid ((size + block.x - 1) / block.x, 1);
    printf("grid: %d, block: %d\n", grid.x, block.x);

    //allocate memory on host
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x*sizeof(int));
    int *tmp = (int *) malloc(bytes);

    //initialize the array
    for(int i = 0; i < size; ++i){
        //maks off high 2 bytes to force max number to 255
        h_idata[i] = (int) (rand() & 0xFF);
    }
    memcpy(tmp, h_idata, bytes);

    size_t iStart, iElaps;
    int gpu_sum=0;

    //allocate memory on GPU
    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x*sizeof(int));

    //cpu reduction
    iStart = cpuSecond();
    int cpu_sum = recursiveReduce(tmp, size);
    iElaps = cpuSecond() - iStart;
    // iElaps = static_cast<double>(end_time.QuadPart - start_time.QuadPart) / frequency.QuadPart / 1000;
    // iElaps = end_time - start_time;
    
    printf("CPU reduce elapsed %d ms cpu_sum: %d\n", iElaps, cpu_sum);

    //warmup
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU warmup elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x, block.x);

    //kernel 1: reduceNeighbored
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU reduceNeighbored elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x, block.x);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x/8*sizeof(int), hipMemcpyDeviceToHost);


    //kernel 2: reduceNeighboredLess
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU reduceNeighbored2 elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x, block.x);
    

    //kernel 3: reduceInterleaved
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU reduceInterleaved elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x, block.x);
    hipDeviceSynchronize();

    //kernel 4: reduceUnrolling2
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceUnrolling2<<<grid.x/2, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x/2*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x/2; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU reduceUnrolling2 elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x/2, block.x);
    hipDeviceSynchronize();

    //kernel 5: reduceCompleteUnrollWarps8
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    reduceCompleteUnrollWarps8<<<grid.x/2, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x/2*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for(int i = 0; i < grid.x/2; ++i) 
        gpu_sum += h_odata[i];
    printf("GPU reduceCompleteUnrollWarps8 elapsed %d ms gpu_sum: %d <<<grid %d block %d>>> \n", iElaps, gpu_sum, grid.x/2, block.x);
    hipDeviceSynchronize();


    // gpu_sum = 0;
    // for(int i = 0; i < grid.x/8; ++i) 
    //     gpu_sum += h_odata[i];

    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

    hipDeviceReset();
    //check the reuslt
    bResult = (gpu_sum == cpu_sum);
    if(!bResult)
        printf(">> Test FAILED!\n");
    return EXIT_SUCCESS;
    
    

}
